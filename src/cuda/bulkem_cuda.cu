#include <pthread.h>
#include <stdio.h>
#include <sys/time.h>

#include "../bulkem.h"
#include "cxx11.h"


void *thread(void *void_args)
{
    fit_params *fp = (fit_params *)void_args;
    stream_main(fp);
    return 0;
}

void print_time_elapsed(struct timeval start_time, struct timeval end_time)
{
    double elapsed_time = (1000000.0 * (end_time.tv_sec - start_time.tv_sec) + end_time.tv_usec - start_time.tv_usec) / 1000000.0;
    printf("%f seconds elapsed\n", elapsed_time);
}

int bulkem_cuda(fit_params *fp)
{
    struct timeval start_time, end_time;
    int num_gpus;

    if (hipSuccess != hipGetDeviceCount(&num_gpus)) {
        printf("Failed to query CUDA devices2\n");
        return -4;
    }

    if (fp->verbose)
        printf("There are %d GPUs\n", num_gpus);

    chunk_reset();

    if (num_gpus != 1) {
        // error("We only support a single GPU right now");
        // return ScalarReal(-1000.0);

        // FIXME: return a more informative error message
        return -1;
    }

    // fire up some threads
    // FIXME: you might revise this in light of the multiple initialisations thing (you could just do one dataset at a time but multiple times; better cache locality, perhaps)
    if (fp->verbose)
        printf("Processing %d datasets simultaneously\n", NUM_THREADS);

    pthread_t threads[NUM_THREADS];

    gettimeofday(&start_time, 0);

    // launch threads
    for (unsigned i = 0; i < NUM_THREADS; i++)
    {
        int rc = pthread_create(&threads[i], NULL, thread, fp);

        if (rc)
        {
            // FIXME: do something smarter
            printf("THREAD LAUNCH FAILED rc=%d\n", rc);
            return 1;
        }
    }

    // join threads
    for (unsigned i = 0; i < NUM_THREADS; i++)
    {
        pthread_join(threads[i], NULL);
    }

    gettimeofday(&end_time, 0);
    printf("Processed %i chunks\n", fp->num_datasets);

    printf("cuda parallel: ");
    print_time_elapsed(start_time, end_time);

    return 0; // TODO: return some sort of success code
}


// FIXME: free everything that you malloced on both host and device - this is more important for R
